#include "hip/hip_runtime.h"
#include "sha3.h"
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>
#include <string.h>

static const char *_cudaErrorToString(hipError_t error) {
  switch (error) {
  case hipSuccess:
    return "hipSuccess";

  case hipErrorMissingConfiguration:
    return "hipErrorMissingConfiguration";

  case hipErrorOutOfMemory:
    return "hipErrorOutOfMemory";

  case hipErrorNotInitialized:
    return "hipErrorNotInitialized";

  case hipErrorLaunchFailure:
    return "hipErrorLaunchFailure";

  case hipErrorPriorLaunchFailure:
    return "hipErrorPriorLaunchFailure";

  case hipErrorLaunchTimeOut:
    return "hipErrorLaunchTimeOut";

  case hipErrorLaunchOutOfResources:
    return "hipErrorLaunchOutOfResources";

  case hipErrorInvalidDeviceFunction:
    return "hipErrorInvalidDeviceFunction";

  case hipErrorInvalidConfiguration:
    return "hipErrorInvalidConfiguration";

  case hipErrorInvalidDevice:
    return "hipErrorInvalidDevice";

  case hipErrorInvalidValue:
    return "hipErrorInvalidValue";

  case hipErrorInvalidPitchValue:
    return "hipErrorInvalidPitchValue";

  case hipErrorInvalidSymbol:
    return "hipErrorInvalidSymbol";

  case hipErrorMapFailed:
    return "hipErrorMapFailed";

  case hipErrorUnmapFailed:
    return "hipErrorUnmapFailed";

  case cudaErrorInvalidHostPointer:
    return "cudaErrorInvalidHostPointer";

  case hipErrorInvalidDevicePointer:
    return "hipErrorInvalidDevicePointer";

  case hipErrorInvalidTexture:
    return "hipErrorInvalidTexture";

  case cudaErrorInvalidTextureBinding:
    return "cudaErrorInvalidTextureBinding";

  case hipErrorInvalidChannelDescriptor:
    return "hipErrorInvalidChannelDescriptor";

  case hipErrorInvalidMemcpyDirection:
    return "hipErrorInvalidMemcpyDirection";

  case cudaErrorAddressOfConstant:
    return "cudaErrorAddressOfConstant";

  case cudaErrorTextureFetchFailed:
    return "cudaErrorTextureFetchFailed";

  case cudaErrorTextureNotBound:
    return "cudaErrorTextureNotBound";

  case cudaErrorSynchronizationError:
    return "cudaErrorSynchronizationError";

  case cudaErrorInvalidFilterSetting:
    return "cudaErrorInvalidFilterSetting";

  case cudaErrorInvalidNormSetting:
    return "cudaErrorInvalidNormSetting";

  case cudaErrorMixedDeviceExecution:
    return "cudaErrorMixedDeviceExecution";

  case hipErrorDeinitialized:
    return "hipErrorDeinitialized";

  case hipErrorUnknown:
    return "hipErrorUnknown";

  case cudaErrorNotYetImplemented:
    return "cudaErrorNotYetImplemented";

  case cudaErrorMemoryValueTooLarge:
    return "cudaErrorMemoryValueTooLarge";

  case hipErrorInvalidHandle:
    return "hipErrorInvalidHandle";

  case hipErrorNotReady:
    return "hipErrorNotReady";

  case hipErrorInsufficientDriver:
    return "hipErrorInsufficientDriver";

  case hipErrorSetOnActiveProcess:
    return "hipErrorSetOnActiveProcess";

  case cudaErrorInvalidSurface:
    return "cudaErrorInvalidSurface";

  case hipErrorNoDevice:
    return "hipErrorNoDevice";

  case hipErrorECCNotCorrectable:
    return "hipErrorECCNotCorrectable";

  case hipErrorSharedObjectSymbolNotFound:
    return "hipErrorSharedObjectSymbolNotFound";

  case hipErrorSharedObjectInitFailed:
    return "hipErrorSharedObjectInitFailed";

  case hipErrorUnsupportedLimit:
    return "hipErrorUnsupportedLimit";

  case cudaErrorDuplicateVariableName:
    return "cudaErrorDuplicateVariableName";

  case cudaErrorDuplicateTextureName:
    return "cudaErrorDuplicateTextureName";

  case cudaErrorDuplicateSurfaceName:
    return "cudaErrorDuplicateSurfaceName";

  case cudaErrorDevicesUnavailable:
    return "cudaErrorDevicesUnavailable";

  case hipErrorInvalidImage:
    return "hipErrorInvalidImage";

  case hipErrorNoBinaryForGpu:
    return "hipErrorNoBinaryForGpu";

  case cudaErrorIncompatibleDriverContext:
    return "cudaErrorIncompatibleDriverContext";

  case hipErrorPeerAccessAlreadyEnabled:
    return "hipErrorPeerAccessAlreadyEnabled";

  case hipErrorPeerAccessNotEnabled:
    return "hipErrorPeerAccessNotEnabled";

  case hipErrorContextAlreadyInUse:
    return "hipErrorContextAlreadyInUse";

  case hipErrorProfilerDisabled:
    return "hipErrorProfilerDisabled";

  case hipErrorProfilerNotInitialized:
    return "hipErrorProfilerNotInitialized";

  case hipErrorProfilerAlreadyStarted:
    return "hipErrorProfilerAlreadyStarted";

  case hipErrorProfilerAlreadyStopped:
    return "hipErrorProfilerAlreadyStopped";

  case hipErrorAssert:
    return "hipErrorAssert";

  case cudaErrorTooManyPeers:
    return "cudaErrorTooManyPeers";

  case hipErrorHostMemoryAlreadyRegistered:
    return "hipErrorHostMemoryAlreadyRegistered";

  case hipErrorHostMemoryNotRegistered:
    return "hipErrorHostMemoryNotRegistered";

  case hipErrorOperatingSystem:
    return "hipErrorOperatingSystem";

  case hipErrorPeerAccessUnsupported:
    return "hipErrorPeerAccessUnsupported";

  case cudaErrorLaunchMaxDepthExceeded:
    return "cudaErrorLaunchMaxDepthExceeded";

  case cudaErrorLaunchFileScopedTex:
    return "cudaErrorLaunchFileScopedTex";

  case cudaErrorLaunchFileScopedSurf:
    return "cudaErrorLaunchFileScopedSurf";

  case cudaErrorSyncDepthExceeded:
    return "cudaErrorSyncDepthExceeded";

  case cudaErrorLaunchPendingCountExceeded:
    return "cudaErrorLaunchPendingCountExceeded";

  case cudaErrorNotPermitted:
    return "cudaErrorNotPermitted";

  case hipErrorNotSupported:
    return "hipErrorNotSupported";

  case cudaErrorHardwareStackError:
    return "cudaErrorHardwareStackError";

  case cudaErrorIllegalInstruction:
    return "cudaErrorIllegalInstruction";

  case cudaErrorMisalignedAddress:
    return "cudaErrorMisalignedAddress";

  case cudaErrorInvalidAddressSpace:
    return "cudaErrorInvalidAddressSpace";

  case cudaErrorInvalidPc:
    return "cudaErrorInvalidPc";

  case hipErrorIllegalAddress:
    return "hipErrorIllegalAddress";

  case hipErrorInvalidKernelFile:
    return "hipErrorInvalidKernelFile";

  case hipErrorInvalidGraphicsContext:
    return "hipErrorInvalidGraphicsContext";

  case cudaErrorStartupFailure:
    return "cudaErrorStartupFailure";

  case cudaErrorApiFailureBase:
    return "cudaErrorApiFailureBase";

  case cudaErrorNvlinkUncorrectable:
    return "cudaErrorNvlinkUncorrectable";

  case cudaErrorJitCompilerNotFound:
    return "cudaErrorJitCompilerNotFound";

  case hipErrorCooperativeLaunchTooLarge:
    return "hipErrorCooperativeLaunchTooLarge";
  }

  return "<unknown>";
}

__device__ int memcmp_cu(const void *p1, const void *p2, size_t len) {
  for (size_t i = 0; i < len; i++) {
    uint8_t b1 = ((uint8_t *)p1)[i];
    uint8_t b2 = ((uint8_t *)p2)[i];
    if (b1 < b2) {
      return -1;
    }
    if (b1 > b2) {
      return 1;
    }
  }
  return 0;
}

__device__ int strlen_cu(char *s) {
  int i;
  for (i = 0; s[i] != '\0';) {
    i++;
  }
  return i;
}

__device__ char *reverse(char *str) {
  char tmp, *src, *dst;
  size_t len;
  if (str != NULL) {
    len = strlen_cu(str);
    if (len > 1) {
      src = str;
      dst = src + len - 1;
      while (src < dst) {
        tmp = *src;
        *src++ = *dst;
        *dst-- = tmp;
      }
    }
  }
  return str;
}

__device__ int itoa(int64_t n, char s[]) {
  int i;
  int64_t sign;

  if ((sign = n) < 0) /* record sign */
    n = -n;           /* make n positive */
  i = 0;

  do {                     /* generate digits in reverse order */
    s[i++] = n % 10 + '0'; /* get next digit */
  } while ((n /= 10) > 0); /* delete it */

  if (sign < 0)
    s[i++] = '-';

  s[i] = '\0';
  reverse(s);
  return i;
}

__device__ void debug_print_buf(const void *buf, size_t len) {
  for (int i = 0; i < len; i++) {
    printf("%c", ((char *)buf)[i]);
  }
  printf("\n");
}

__device__ void debug_print_hash(const void *hash) {
  for (int i = 0; i < 32; i++) {
    printf("%02x", ((char *)hash)[i] & 0xFF);
  }
  printf("\n");
}

// called from the gpu kernel
__global__ void do_sha3(const void *first, size_t first_len, const void *last,
                        size_t last_len, int64_t start_nonce, void *target,
                        int64_t *good_nonce, int *hashes) {
  uint8_t hash[32];
  uint8_t nonce_s[20];

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int64_t nonce = start_nonce + (int64_t)index;
  size_t n = (size_t)itoa(nonce, (char *)nonce_s);

  sha3_ctx_t sha3;

  sha3_init_cu(&sha3, 32);
  sha3_update_cu(&sha3, first, first_len);
  sha3_update_cu(&sha3, nonce_s, n);
  sha3_update_cu(&sha3, last, last_len);
  sha3_final_cu(hash, &sha3);

  // atomicAdd(hashes, 1);
#if 0
  if (index == 0) {
    debug_print_buf(first, first_len);
    debug_print_buf(nonce_s, n);
    debug_print_buf(last, last_len);
    debug_print_hash(hash);
    debug_print_hash(target);
  }
#endif

  if (memcmp_cu(hash, target, 32) <= 0) {
#if 0
    debug_print_buf(first, first_len);
    debug_print_buf(nonce_s, n);
    debug_print_buf(last, last_len);
    debug_print_hash(target);
    debug_print_hash((char *)hash);
#endif
    // found a solution. not thread-safe but a race is very unlikely
    *good_nonce = nonce;
  }
}

struct miner_state {
  void *first_cu, *last_cu, *target_cu;
  size_t first_len, last_len;
  int num_blocks, block_size, max_threads;
  int64_t *nonce_cu;
  int *hashes_cu;
};

static struct miner_state *states = 0;

extern "C" {

// called on startup
int cuda_init() {
  int device_count = -1;
  hipError_t error = hipGetDeviceCount(&device_count);
  if (error != hipSuccess) {
    printf("hipGetDeviceCount: %s\n", _cudaErrorToString(error));
    return -1;
  }
  if (device_count <= 0) {
    return -1;
  }

  states = new struct miner_state[device_count];

  for (int i = 0; i < device_count; i++) {
    hipDeviceProp_t props;
    error = hipGetDeviceProperties(&props, i);
    if (error != hipSuccess) {
      printf("hipGetDeviceProperties: %s\n", _cudaErrorToString(error));
      return -1;
    }

    states[i].max_threads =
        props.maxThreadsPerMultiProcessor * props.multiProcessorCount;
    states[i].block_size = props.warpSize;
    states[i].num_blocks = states[i].max_threads / states[i].block_size;

    error = hipSetDevice(i);
    if (error != hipSuccess) {
      printf("hipSetDevice: %s\n", _cudaErrorToString(error));
      return -1;
    }

    // allocate memory used on device
    hipMalloc(&states[i].first_cu, 512);
    hipMalloc(&states[i].last_cu, 512);
    hipMalloc(&states[i].target_cu, 32);
    hipMalloc(&states[i].nonce_cu, sizeof(int64_t));
    hipMalloc(&states[i].hashes_cu, sizeof(int));

    hipMemset(states[i].hashes_cu, 0, sizeof(int));
    hipMemset(states[i].nonce_cu, 0x7F, sizeof(int64_t));
    hipMemset(states[i].nonce_cu, 0xFF, sizeof(int64_t) - 1);
  }

  return device_count;
}

// called after updating the block header
int miner_update(int miner_num, const void *first, size_t first_len,
                 const void *last, size_t last_len, const void *target) {
  hipSetDevice(miner_num);

  // copy the first part of the header
  states[miner_num].first_len = first_len;
  hipMemcpy(states[miner_num].first_cu, first, first_len,
             hipMemcpyHostToDevice);

  // copy the end part of the header
  states[miner_num].last_len = last_len;
  hipMemcpy(states[miner_num].last_cu, last, last_len, hipMemcpyHostToDevice);

  // copy the target
  hipMemcpy(states[miner_num].target_cu, target, 32, hipMemcpyHostToDevice);

  // clear nonce
  hipMemset(states[miner_num].nonce_cu, 0x7F, sizeof(int64_t));
  hipMemset(states[miner_num].nonce_cu, 0xFF, sizeof(int64_t) - 1);

  return states[miner_num].num_blocks * states[miner_num].block_size;
}

// called in a loop until solved
// returns a solving nonce if found; otherwise 0x7FFFFFFFFFFFFFFF
int64_t miner_mine(int miner_num, int64_t start_nonce) {
  hipSetDevice(miner_num);
  int64_t nonce;
  hipMemset(states[miner_num].hashes_cu, 0, sizeof(int));
  int num_blocks = states[miner_num].num_blocks;
  int block_size = states[miner_num].block_size;
  do_sha3<<<num_blocks, block_size>>>(
      states[miner_num].first_cu, states[miner_num].first_len,
      states[miner_num].last_cu, states[miner_num].last_len, start_nonce,
      states[miner_num].target_cu, states[miner_num].nonce_cu,
      states[miner_num].hashes_cu);
  hipDeviceSynchronize();
  hipMemcpy(&nonce, states[miner_num].nonce_cu, sizeof(int64_t),
             hipMemcpyDeviceToHost);
  return nonce;
}
}
