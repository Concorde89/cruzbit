#include "hip/hip_runtime.h"
#include "sha3.h"
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>
#include <string.h>

inline int _ConvertSMVer2Cores(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine the #
  // of cores per SM
  typedef struct {
    int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM
            // minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
      {0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
      {0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
      {0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
      {0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
      {0x52, 128}, // Maxwell Generation (SM 5.2) GM20x class
      {0x53, 128}, // Maxwell Generation (SM 5.3) GM20x class
      {0x60, 64},  // Pascal Generation (SM 6.0) GP100 class
      {0x61, 128}, // Pascal Generation (SM 6.1) GP10x class
      {0x62, 128}, // Pascal Generation (SM 6.2) GP10x class
      {0x70, 64},  // Volta Generation (SM 7.0) GV100 class
      {-1, -1}};
  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }
    index++;
  }

  // If we don't find the values, we default use the previous one to run
  // properly
  printf(
      "MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

__device__ int memcmp_cu(const void *p1, const void *p2, size_t len) {
  for (size_t i = 0; i < len; i++) {
    uint8_t b1 = ((uint8_t *)p1)[i];
    uint8_t b2 = ((uint8_t *)p2)[i];
    if (b1 < b2) {
      return -1;
    }
    if (b1 > b2) {
      return 1;
    }
  }
  return 0;
}

__device__ int strlen_cu(char *s) {
  int i;
  for (i = 0; s[i] != '\0';) {
    i++;
  }
  return i;
}

__device__ char *reverse(char *str) {
  char tmp, *src, *dst;
  size_t len;
  if (str != NULL) {
    len = strlen_cu(str);
    if (len > 1) {
      src = str;
      dst = src + len - 1;
      while (src < dst) {
        tmp = *src;
        *src++ = *dst;
        *dst-- = tmp;
      }
    }
  }
  return str;
}

__device__ int itoa(int64_t n, char s[]) {
  int i;
  int64_t sign;

  if ((sign = n) < 0) /* record sign */
    n = -n;           /* make n positive */
  i = 0;

  do {                     /* generate digits in reverse order */
    s[i++] = n % 10 + '0'; /* get next digit */
  } while ((n /= 10) > 0); /* delete it */

  if (sign < 0)
    s[i++] = '-';

  s[i] = '\0';
  reverse(s);
  return i;
}

__device__ void debug_print_buf(const void *buf, size_t len) {
  for (int i = 0; i < len; i++) {
    printf("%c", ((char *)buf)[i]);
  }
  printf("\n");
}

__device__ void debug_print_hash(const void *hash) {
  for (int i = 0; i < 32; i++) {
    printf("%02x", ((char *)hash)[i] & 0xFF);
  }
  printf("\n");
}

// called from the gpu kernel
__global__ void do_sha3(const void *first, size_t first_len, const void *last,
                        size_t last_len, int64_t start_nonce, void *target,
                        int64_t *good_nonce, int *hashes) {
  uint8_t hash[32];
  uint8_t nonce_s[20];

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int64_t nonce = start_nonce + (int64_t)index;
  size_t n = (size_t)itoa(nonce, (char *)nonce_s);

  sha3_ctx_t sha3;

  sha3_init_cu(&sha3, 32);
  sha3_update_cu(&sha3, first, first_len);
  sha3_update_cu(&sha3, nonce_s, n);
  sha3_update_cu(&sha3, last, last_len);
  sha3_final_cu(hash, &sha3);

  // atomicAdd(hashes, 1);
#if 0
  if (index == 0) {
    debug_print_buf(first, first_len);
    debug_print_buf(nonce_s, n);
    debug_print_buf(last, last_len);
    debug_print_hash(hash);
    debug_print_hash(target);
  }
#endif

  if (memcmp_cu(hash, target, 32) <= 0) {
#if 0
    debug_print_buf(first, first_len);
    debug_print_buf(nonce_s, n);
    debug_print_buf(last, last_len);
    debug_print_hash(target);
    debug_print_hash((char *)hash);
#endif
    // found a solution. not thread-safe but a race is very unlikely
    *good_nonce = nonce;
  }
}

struct miner_state {
  void *first_cu, *last_cu, *target_cu;
  size_t first_len, last_len;
  int num_blocks, block_size, max_threads;
  int64_t *nonce_cu;
  int *hashes_cu;
};

static struct miner_state *states = 0;

extern "C" {

// called on startup
int cuda_init() {
  int device_count = -1;
  hipGetDeviceCount(&device_count);
  if (device_count <= 0) {
    return -1;
  }

  states = new struct miner_state[device_count];

  for (int i = 0; i < device_count; i++) {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, i);
    int cores = props.major == 9999 && props.minor == 9999
                    ? 1
                    : _ConvertSMVer2Cores(props.major, props.minor);
    cores *= props.multiProcessorCount;

    states[i].max_threads =
        props.maxThreadsPerMultiProcessor * props.multiProcessorCount;
    states[i].block_size = props.warpSize;
    states[i].num_blocks = states[i].max_threads / states[i].block_size;

    // allocate memory used on device
    hipMalloc(&states[i].first_cu, 512);
    hipMalloc(&states[i].last_cu, 512);
    hipMalloc(&states[i].target_cu, 32);
    hipMalloc(&states[i].nonce_cu, sizeof(int64_t));
    hipMalloc(&states[i].hashes_cu, sizeof(int));

    hipMemset(states[i].hashes_cu, 0, sizeof(int));
    hipMemset(states[i].nonce_cu, 0x7F, sizeof(int64_t));
    hipMemset(states[i].nonce_cu, 0xFF, sizeof(int64_t) - 1);
  }

  return device_count;
}

// called after updating the block header
int miner_update(int miner_num, const void *first, size_t first_len,
                 const void *last, size_t last_len, const void *target) {
  hipSetDevice(miner_num);

  // copy the first part of the header
  states[miner_num].first_len = first_len;
  hipMemcpy(states[miner_num].first_cu, first, first_len,
             hipMemcpyHostToDevice);

  // copy the end part of the header
  states[miner_num].last_len = last_len;
  hipMemcpy(states[miner_num].last_cu, last, last_len, hipMemcpyHostToDevice);

  // copy the target
  hipMemcpy(states[miner_num].target_cu, target, 32, hipMemcpyHostToDevice);

  // clear nonce
  hipMemset(states[miner_num].nonce_cu, 0x7F, sizeof(int64_t));
  hipMemset(states[miner_num].nonce_cu, 0xFF, sizeof(int64_t) - 1);

  return states[miner_num].num_blocks * states[miner_num].block_size;
}

// called in a loop until solved
// returns a solving nonce if found; otherwise 0x7FFFFFFFFFFFFFFF
int64_t miner_mine(int miner_num, int64_t start_nonce) {
  hipSetDevice(miner_num);
  int64_t nonce;
  hipMemset(states[miner_num].hashes_cu, 0, sizeof(int));
  int num_blocks = states[miner_num].num_blocks;
  int block_size = states[miner_num].block_size;
  do_sha3<<<num_blocks, block_size>>>(
      states[miner_num].first_cu, states[miner_num].first_len,
      states[miner_num].last_cu, states[miner_num].last_len, start_nonce,
      states[miner_num].target_cu, states[miner_num].nonce_cu,
      states[miner_num].hashes_cu);
  hipDeviceSynchronize();
  hipMemcpy(&nonce, states[miner_num].nonce_cu, sizeof(int64_t),
             hipMemcpyDeviceToHost);
  return nonce;
}
}
